
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define min(a,b) (a<b?a:b)

#define threadsPerBlock  256
#define N  33 * 1024
#define blocksPerGrid  min(32, (N+threadsPerBlock-1)/threadsPerBlock)

__global__ void dot(float *a, float *b, float *c) {

    //calculate thread id combining the block and thread indices to get global offset into the input arrays
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    //shared memory for each block, which means each block has a copy of the memory.
    //and the index of the cache is just the thread index in each block
    __shared__ float cache[threadsPerBlock];
    int cache_index = threadIdx.x;

    int i;
    float tmp = 0;
    //each thread multiplies a pair of corresponding entries, and then every thread moves on to its next pare.
    //the threads increment their indices by the total number of threads to easure we don't miss any elements and don't multiply a pair twice
    while (tid < N) {
        tmp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    //in each block, store the sum of pairs from each thread
    cache[cache_index] = tmp;
    //sync threads in the block before we sum all the values resulted from each thread.
    __syncthreads();

    //for reductions, threads Per block must be power of 2 because of the following reduction
    i = blockDim.x/2;
    while (i!=0) {
        if(cache_index < i) {
            cache[cache_index] += cache[cache_index+i];
            //__syncthreads();   THIS IS NOT ALLOWED and GPU will not work!!!!
        }
        //sync threads  after each iteration of reduction
        //notice that the "__syncthreads" cannot be placed in the above "if" block
        //because cuda architecture guarantees that no thread will advance to an instruction beyond the __syncthreads() until every
        //thread in the block has executed the "syncthreads", however, if the "__syncthreads" is placed into a divergent branch,
        //some threads block will never go to the branch and hardware will simply continue to wait for these threads, forever.
         __syncthreads();
         i/=2;
    }

    //Use one thread in each block to write the results of each block to the global memory
    //here "c" gather each block's sum results, since there is not many blocks, we don't leverage GPU to complete the final results
    //and use CPU to compute this part
    if (cache_index == 0 ) {
        c[blockIdx.x] = cache[0];
    }

}

int main(void) {
    float *a, *b, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    int i;
    float sum;

    //allocate memory for array a, b and partial_c on CPU side
    a = (float *)malloc(N * sizeof(float));
    b = (float *)malloc(N * sizeof(float));
    partial_c = (float *)malloc(blocksPerGrid * sizeof(float));

    //initialize a and b in CPU
    for (i=0;i<N; i++){
        a[i] = i;
        b[i] = i*2;
    }

    //malloc memory on GPU for array a, array b and partial results for each block
    hipMalloc((void **) &dev_a, N * sizeof(float) );
    hipMalloc((void **) &dev_b, N * sizeof(float) );
    hipMalloc((void **) &dev_partial_c, blocksPerGrid * sizeof(float) );

    //copy memory from host to device
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_partial_c, partial_c,  blocksPerGrid * sizeof(float), hipMemcpyHostToDevice);

    //call the kernel
    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);
    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

    for (i = 0; i< blocksPerGrid; i++) {
        sum += partial_c[i];
    }

    //verify whether the result is correct
    #define sum_squares(x) (x*(x+1)*(2*x+1)/6)
    float expect_value = 2 * sum_squares((float)(N-1));

    printf("does the gpu value %.6g = %.6g\n", sum, expect_value);

    //free memory on GPU side
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);

    //free memory on CPU side
    free(a);
    free(b);
    free(partial_c);

    return 0;
}
